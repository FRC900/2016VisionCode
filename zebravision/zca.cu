#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include "opencv2_3_shim.hpp"

#include "cuda_utils.hpp"

using std::cout;
using std::endl;
#if CV_MAJOR_VERSION == 2
using cv::gpu::PtrStepSz;
#elif CV_MAJOR_VERSION == 3
using cv::cuda::PtrStepSz;
#endif

// Take the output of the ZCA matrix mul - that will
// be a matrix. Each image is a row, each row is the pixels
// in BGRBGRBGR.. order
// Convert that to a flat 1-D array as expected by the neural
// net input stages
__global__ void unflatten_kernel(const PtrStepSz<float> input,
								 const size_t rows,
								 const size_t cols,
							 	 float *output)
{
	// 2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	// Each image it its own zIndex
	const int zIndex = blockIdx.z * blockDim.z + threadIdx.z;

	//Only valid threads perform memory I/O
	if((xIndex < cols) && (yIndex < rows))
	{
		// yIndex * cols = number of floats per complete
		// filled row
		// add xIndex to get to the correct location in this row
		// Multiply by three to account for R, G, B float values
		//   per col in the input images
		const int flatIdxX = 3*(yIndex * cols + xIndex);
		const float blue  = input(zIndex, flatIdxX + 0);
		const float green = input(zIndex, flatIdxX + 1);
		const float red	  = input(zIndex, flatIdxX + 2);

		// Convert to flat 1-D representation
		// order is [image][color channel][row][col]
		const int chanDist = rows * cols;
		const int idx = zIndex * 3 * chanDist + // 3 channels of row*col pixels per image
			            yIndex * cols +         // select correct row   
						xIndex;                 // and the column in that row

		output[idx]                = blue;      // all the blue comes first
		output[idx +     chanDist] = green;     // then the green 
		output[idx + 2 * chanDist] = red;       // then the red from a given image
	}
}

// Math to add two intermediate steps of mean & stddev 
// See http://www.johndcook.com/blog/skewness_kurtosis/
__device__ void combine_running_totals(float &M1_1, const float M1_2, float &M2_1, const float M2_2, unsigned int &n_1, const unsigned int n_2)
{
	unsigned int combined_n = n_1 + n_2;

	const float delta  = M1_2 - M1_1;
	const float delta2 = delta * delta;

	float combined_M1 = (n_1 * M1_1 + n_2 * M1_2) / combined_n;
	float combined_M2 = M2_1 + M2_2 + delta2 * n_1 * n_2 / combined_n;

	n_1  = combined_n;
	M1_1 = combined_M1;
	M2_1 = combined_M2;
}

// For each input image, calculate the mean and stddev
// of each color channel in each image.  Then, for each
// pixel in a given image, apply global contrast normalization
// to the image - subtract the mean and divide by the stddev
// of the color channel of that image.
// input is an array of images, output is a 2d matrix where
// each image has been flattened into a single row
__global__ void mean_stddev_reduction_kernel(const PtrStepSz<float> *input,
												   PtrStepSz<float> output)
{
	// Thread index within block - used for addressing smem below
	const unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

	// Shared memory per channel per thread = 1 long, 2 floats.
	// So a 3 channel image needs 3 longs and 6 floats
	// Thread blocks are up to 24x24 images, one thread per pixel
	// TODO : fixme for variable sized thread blocks
	__shared__ float M1[32*32*3];
	__shared__ float M2[32*32*3];
	__shared__ unsigned int n[32*32*3];

	// 2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	// Each image it its own zIndex
	const int zIndex = blockIdx.z * blockDim.z + threadIdx.z;

	// Only valid threads perform memory I/O
	if((xIndex < input[zIndex].cols) && (yIndex < input[zIndex].rows))
	{
		// xIndex * 3 since col has a blue green and red component
		const float blue  = input[zIndex](yIndex, 3*xIndex);
		const float green = input[zIndex](yIndex, 3*xIndex + 1);
		const float red	  = input[zIndex](yIndex, 3*xIndex + 2);

		// Initialize running average
		M1[tid * 3]     = blue;
		M1[tid * 3 + 1] = green;
		M1[tid * 3 + 2] = red;

		// Initialize pixel count
		n[tid * 3]     = 1;
		n[tid * 3 + 1] = 1;
		n[tid * 3 + 2] = 1;
	}
	else
	{
		// This thread has nothing to contribute
		// to the final result
		n[tid * 3]     = 0;
		n[tid * 3 + 1] = 0;
		n[tid * 3 + 2] = 0;
	}

	M2[tid * 3]     = 0;
	M2[tid * 3 + 1] = 0;
	M2[tid * 3 + 2] = 0;
	
    __syncthreads();

    // do reduction in shared mem
	// For each thread, combine the results from 2 threads
	// down into one. Each pass through the loop eliminates
	// half of the partial results, eventually ending up
	// with just one final result per block
    for (unsigned int s = (blockDim.x * blockDim.y) / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
			for (int i = 0; i < 3; i++)
			{
				// Blue, green, red = 3 entries per shared mem array
				const int i1 = 3 * tid + i;
				const int i2 = 3 * (tid + s) + i;
				if (n[i2])
					combine_running_totals(M1[i1], M1[i2], M2[i1], M2[i2], n[i1], n[i2]);
n[i2] = 0;
			}
        }
        __syncthreads();
    }

    // Update M1[0-2] and M2[0-2] with the 
    // mean and stddev of the B, G, R pixels
    if (tid < 3)
	{
		// M1 is the mean already - nothing extra needed
		// calculate stddev from M2 and n
		M2[tid] = sqrt(M2[tid] / n[tid]);
	}
	__syncthreads();

	// Apply global contrast normalization to
	// each input image.
	// For each channel in each image, the mean and stddev has
	// already been calculated
	// For each channel in each pixel, subtract the mean and divide by the stddev
	// Insure only valid threads perform memory I/O
	// If the x/y index for this thread is beyond the
	// number of cols/rows, do nothing
	if((xIndex < input[zIndex].cols) && (yIndex < input[zIndex].rows))
	{
		// xIndex * 3 since col has a blue green and red component
		float blue	= input[zIndex](yIndex, 3 * xIndex);
		float green	= input[zIndex](yIndex, 3 * xIndex + 1);
		float red	= input[zIndex](yIndex, 3 * xIndex + 2);

		blue  = (blue  - M1[0]) / M2[0];
		green = (green - M1[1]) / M2[1];
		red   = (red   - M1[2]) / M2[2];

		// yIndex * input[0].cols = number of floats per complete
		// filled row
		// add xIndex to get to the correct location in this row
		// Multiply by three to account for R, G, B float values
		//   per col in the input images
		const int flatIdxX = 3 * (yIndex * input[zIndex].cols + xIndex);
		output(zIndex, flatIdxX + 0) = blue;
		output(zIndex, flatIdxX + 1) = green;
		output(zIndex, flatIdxX + 2) = red;
	}
}

__host__ void cudaZCATransform(const std::vector<GpuMat> &input, 
		const GpuMat &weights, 
		PtrStepSz<float> *dPssIn,
		GpuMat &dFlattenedImages,
		GpuMat &zcaOut,
		float *output)
{
	// Create array of PtrStepSz entries corresponding to
	// each GPU mat in input. Copy it to device memory
	PtrStepSz<float> hPssIn[input.size()];
	for (size_t i = 0; i < input.size(); ++i)
		hPssIn[i] = input[i];
	cudaSafeCall(hipMemcpy(dPssIn, hPssIn, input.size() * sizeof(PtrStepSz<float>), hipMemcpyHostToDevice), "hipMemcpy dPssIn");

	// Each block is one image
	// Set the block size to the smallest power
	// of two large enough to hold an image
	dim3 block;
	if (input[0].cols == 12)
		block = dim3(16, 16);
	else
		block = dim3(32, 32);

	// z dimension is number of images
	const dim3 grid(1, 1, input.size());

	// Todo : do this once in ZCA constructor
	// Create a CUDA stream. This lets us queue up a number of
	// cuda calls back to back and then later check to see
	// that they all finished
	hipStream_t stream;
	cudaSafeCall(hipStreamCreate(&stream), "ZCA hipStreamCreate");

	// Todo : do this once in ZCA constructor
    hipblasHandle_t handle;
    cublasSafeCall(hipblasCreate(&handle), "hipblasCreate");
    cublasSafeCall(hipblasSetStream(handle, stream), "hipblasSetStream");

    cublasSafeCall(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST), "hipblasSetPointerMode");
    const float alpha = 1.0;
    const float beta = 0.0;

	//Launch the first reduction kernel
	// this will output an array of intermediate values
	// in M1 (running average) and M2 (variance * number 
	// of values seen). n is number of values corresponding
	// to each M1 and M2 value.
	mean_stddev_reduction_kernel<<<grid,block,0,stream>>>(dPssIn, dFlattenedImages);
	//cudaSafeCall(hipStreamSynchronize(stream),"ZCA hipStreamSynchronize failed");


	// Todo : do this once in ZCA constructor
	zcaOut.create(dFlattenedImages.size(), dFlattenedImages.type());

	// Multiply images by weights to get the ZCA-whitened output
	cublasSafeCall(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, weights.cols, dFlattenedImages.rows, weights.rows,
		&alpha,
		weights.ptr<float>(), static_cast<int>(weights.step / sizeof(float)),
		dFlattenedImages.ptr<float>(), static_cast<int>(dFlattenedImages.step / sizeof(float)),
		&beta,
		zcaOut.ptr<float>(), static_cast<int>(zcaOut.step / sizeof(float))),
		"hipblasSgemm"	);

	// Copy to output buffer in the order expected by
	// neural net input
	unflatten_kernel<<<grid,block,0,stream>>>(zcaOut, input[0].rows, input[0].cols, output);

	cudaSafeCall(hipStreamSynchronize(stream),"ZCA hipStreamSynchronize failed");
	cublasSafeCall(hipblasDestroy(handle), "hipblasDestroy");
	cudaSafeCall(hipStreamDestroy(stream), "ZCA hipStreamDestroy failed");
}
