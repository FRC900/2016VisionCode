#include "hip/hip_runtime.h"
#include "cuda_utils.hpp"
#include "opencv2_3_shim.hpp"

#if CV_MAJOR_VERSION == 2
using cv::gpu::PtrStepSz;
#elif CV_MAJOR_VERSION == 3
using cv::cuda::PtrStepSz;
#endif

// Given a depth map in input, see if any value is
// in the range between depthMin and depthMax.  If
// so, set answer to true. If all pixels fall outside
// the range, set answer to false.
__global__ void depth_threshold_kernel(const PtrStepSz<float> input,
									   const float depthMin,
									   const float depthMax,
									   bool *answer)
{
	// Thread index within block - used for addressing smem below
	const unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

	__shared__ bool inRange[32*32*3];

	// 2D pixel index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	// Only valid threads perform memory I/O
	if((xIndex < input.cols) && (yIndex < input.rows))
	{
		// Be conservative here - if any of the depth values in the 
		// target rect are in the expected range, consider the entire 
		// rect in range.  Also say that it is in range if any of the 
		// depth values are negative (i.e. no depth info for those pixels)
		const float depth = input(yIndex, xIndex);
		if ((depth <= 0) || ((depth > depthMin) && (depth < depthMax)))
			inRange[tid] = true;
		else
			inRange[tid] = false;
	}
	else
	{
		// Set values outside the range of the image
		// to false. This will make them ignored in
		// the reduction down to a single compare value
		inRange[tid] = false;
	}

	// Let all threads finish the compare and put
	// their results in shared mem
    __syncthreads();

    // do reduction in shared mem
	// For each thread, combine the results from 2 threads
	// down into one. Each pass through the loop eliminates
	// half of the partial results, eventually ending up
	// with just one final result per block
    for (unsigned int s = (blockDim.x * blockDim.y) / 2; s > 0; s >>= 1)
    {
		if (inRange[0] == true)
		{
			if (tid == 0)
				*answer = inRange[0];
			return;
		}
		// Basically just propagate any true values
		// down to thread 0 - only return false
		// if the entire set of compares was false
        if ((tid < s) && inRange[tid + s])
			inRange[tid] = true;
        __syncthreads();
    }

    if (tid == 0)
		*answer = inRange[0];
}

__host__ bool cudaDepthThreshold(const GpuMat &image, const float depthMin, const float &depthMax)
{
	bool *dResult;
	bool  hResult;

	cudaSafeCall(hipMalloc(&dResult, sizeof(bool)), "hipMalloc threshold result");

	// Each block is one image
	// Set the block size to the smallest power
	// of two large enough to hold an image
	const dim3 block(16, 16);

	// only do 1 image at a time for now
	const dim3 grid(1, 1);

	depth_threshold_kernel<<<grid, block>>>(image, depthMin, depthMax, dResult);
	cudaSafeCall(hipDeviceSynchronize(), "depthThreshold hipDeviceSynchronize failed");

	cudaSafeCall(hipMemcpy(&hResult, dResult, sizeof(bool), hipMemcpyDeviceToHost), "hipMemcpy depth result");
	cudaSafeCall(hipFree(dResult), "depthThreshold hipFree");

	return hResult;
}
